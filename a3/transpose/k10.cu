
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
    int pitch_a, int pitch_b, int pitch_c,
    int n, int m, int p )
{
    int tx = threadIdx.x;
    int bx = blockDim.x;
    int i = blockIdx.x*bx*2 + threadIdx.x;
    int j = blockIdx.y*2;
    __shared__ float cc0[512], cc1[512]; 
    float sum0 = 0.0, sum1=0.0, sum2 =0.0, sum3=0.0;

    for( int ks = 0; ks < p; ks += bx ){
        cc0[tx] = c[(ks+tx) * pitch_c + j];
        cc1[tx] = c[(ks+tx) * pitch_c + j+1];
        __syncthreads();
        for( int k = ks; k < ks+bx; ++k ) {
            sum0 += b[i+pitch_b*k] * cc0[k-ks];
            sum1 += b[i+bx+pitch_b*k] * cc0[k-ks];
            sum2 += b[i+pitch_b*k] * cc1[k-ks];
            sum3 += b[i+bx+pitch_b*k] * cc1[k-ks];
        }
        __syncthreads();
    }

    a[j+pitch_a*i] = sum0;
    a[j+pitch_a*(i+bx)] = sum1;
    a[j+1+pitch_a*i] = sum2;
    a[j+1+pitch_a*(i+bx)] = sum3;
}

