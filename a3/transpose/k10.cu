
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
int pitch_a, int pitch_b, int pitch_c,
int n, int m, int p )
{
   int tx = threadIdx.x;
   int i = blockIdx.x*64 + threadIdx.x;
   int j = blockIdx.y*2;
   __shared__ float cc0[32], cc1[32]; 
   float sum0 = 0.0, sum1=0.0, sum2 =0.0, sum3=0.0;

   for( int ks = 0; ks < p; ks += 32 ){
      cc0[tx] = c[(ks+tx) * pitch_c + j];
      cc1[tx] = c[(ks+tx) * pitch_c + j+1];
      __syncthreads();
      for( int k = ks; k < ks+32; ++k ) {
        sum0 += b[i+pitch_b*k] * cc0[k-ks];
	sum1 += b[i+32+pitch_b*k] * cc0[k-ks];
	sum2 += b[i+pitch_b*k] * cc1[k-ks];
	sum3 += b[i+32+pitch_b*k] * cc1[k-ks];
      }
      __syncthreads();
    }
   
   a[j+pitch_a*i] = sum0;
   a[j+pitch_a*(i+32)] = sum1;
   a[j+1+pitch_a*i] = sum2;
   a[j+1+pitch_a*(i+32)] = sum3;
}

