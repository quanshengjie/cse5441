
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
int pitch_a, int pitch_b, int pitch_c,
int n, int m, int p )
{
   int tx = threadIdx.x;
   int i = blockIdx.x*32 + threadIdx.x;
   int j = blockIdx.y;
   __shared__ float cc[32]; 
   float sum = 0.0;
   
   
   for( int ks = 0; ks < p; ks += 32 ){
      cc[tx] = c[(ks+tx) * pitch_c + j];
      __syncthreads();
      for( int k = ks; k < ks+32; ++k )
        sum += b[i+pitch_b*k] * cc[k-ks];
    }
   
   a[j+pitch_a*i] = sum;
}

