
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
int pitch_a, int pitch_b, int pitch_c,
int n, int m, int p )
{
   int tx = threadIdx.x;
   int i = blockIdx.x*64 + threadIdx.x;
   int j = blockIdx.y;
   __shared__ float cc[32]; 
   float sum0 = 0.0, sum1=0.0;

   for( int ks = 0; ks < p; ks += 32 ){
      cc[tx] = c[(ks+tx) * pitch_c + j];
      __syncthreads();
      for( int k = ks; k < ks+32; ++k ) {
        sum0 += b[i+pitch_b*k] * cc[k-ks];
	sum1 += b[i+32+pitch_b*k] * cc[k-ks];
      }
      __syncthreads();
    }
   
   a[j+pitch_a*i] = sum0;
   a[j+pitch_a*(i+32)] = sum1;
}

