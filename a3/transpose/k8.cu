
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
    int pitch_a, int pitch_b, int pitch_c,
    int n, int m, int p )
{
    int tx = threadIdx.x;
    int bx = blockDim.x;
    int i = blockIdx.x*bx*2 + threadIdx.x;
    int j = blockIdx.y;
    __shared__ float cc[512]; 
    float sum0 = 0.0, sum1=0.0;

    for( int ks = 0; ks < p; ks += bx ){
        cc[tx] = c[(ks+tx) * pitch_c + j];
        __syncthreads();
        for( int k = ks; k < ks+bx; ++k ) {
            sum0 += b[i+pitch_b*k] * cc[k-ks];
            sum1 += b[i+bx+pitch_b*k] * cc[k-ks];
        }
        __syncthreads();
    }

    a[j+pitch_a*i] = sum0;
    a[j+pitch_a*(i+bx)] = sum1;
}

