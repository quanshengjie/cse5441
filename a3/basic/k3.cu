
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
  int pitch_a, int pitch_b, int pitch_c,
  int n, int m, int p )
{
    int tx = threadIdx.x;
    int i = blockIdx.x*64 + tx;
    int j = blockIdx.y;
    __shared__ float cb[32];

    float sum0 = 0.0, sum1 = 0.0;
    for( int ks = 0; ks < p; ks += 32 ){
      cb[tx] = c[ks+tx+pitch_c*j];
      __syncthreads();
      for( int k = ks; k < ks+32; ++k ){
        sum0 += b[i+pitch_b*k] * cb[k-ks];
        sum1 += b[i+32+pitch_b*k] * cb[k-ks];
      }
      __syncthreads();
    }
    a[i+pitch_a*j] = sum0;
    a[i+32+pitch_a*j] = sum1;
}
