
#include <hip/hip_runtime.h>
extern "C" __global__ void
mmkernel( float* a, float* b, float* c,
  int pitch_a, int pitch_b, int pitch_c,
  int n, int m, int p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y;

    float sum = 0.0;
    for( int k = 0; k < p; ++k )
      sum += b[i+pitch_b*k] * c[k+pitch_c*j];
    a[i+pitch_a*j] = sum;
}
